#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <string.h>
#define THREADS_PER_BLOCK 256




// The same function as in the host file , only with proper decalration

__device__ void cudaUpdateSymbolArray(char symbol, int *arr)
{ // Function update the symbol array ,and add 1 to the proper index
    if (symbol == '$')
        arr[0]++;
    else if (symbol == '%')
        arr[1]++;
    else if (symbol == '#')
        arr[2]++;
    else
        arr[3]++;
}




// The same function as in the host file , only with proper decalration

__device__ int cudaAlignmentScore(int *weights, int *symbolCount)
{ // calculates the alignment score of two strings according to given equation
    int score, positiveScore, negetiveScore;
    positiveScore = weights[0] * symbolCount[0];
    negetiveScore = weights[1] * symbolCount[1] + weights[2] * symbolCount[2] + weights[3] * symbolCount[3];
    score = positiveScore - negetiveScore;
    return score; // return the ultimate score of the to strings
}







__global__ void kernelFunc(char *seq1, char *seq2, char *matrix, int *weights, int *scoreArr, int num_offsets, int seq2_len, int num_threads){
    // the kernel function , run on the gpu each thread will calculate aligmnet score and save it in the device score array

    int tid = blockDim.x * blockIdx.x + threadIdx.x; // calculate the thread index in the grid 
    if(tid > num_threads) // make sure we do not exceeded from the grid size , only relevent for the last block 
        return;

    int hyphen,offset; // initialize variables
    char result;

    hyphen = tid /num_offsets;      // calculate the values of the offset and the mutant for current thread
    offset = tid % num_offsets;

    int symbolCount[4] = {0};        // Hold for each symbol the number of appearences in comparision ,for calculate score

    for (int k = 0; k < seq2_len; k++) // go over str2 and compare to seq1 char by char and calculate aligment score(same logic as in the host)
                                        // but only with the proper functions for the device 
    {
        if (k >= hyphen)
        {
            result = matrix[(seq1[k + offset + 1] - 'A')*MATRIXSIZE+ seq2[k] - 'A'];
            cudaUpdateSymbolArray(result, symbolCount);
        }
        else
        {
            result = matrix[(seq1[k + offset] - 'A')*MATRIXSIZE + seq2[k] - 'A'];
            cudaUpdateSymbolArray(result, symbolCount); 
        }
    }
    scoreArr[tid] = cudaAlignmentScore(weights, symbolCount); 
   
}






void findMaxScoreCuda(char *seq1, char *seq2, char matrix[MATRIXSIZE][MATRIXSIZE], int *weights, int *scoreArr, int start_hyp, int end_hyp){
    char* dev_seq1,*dev_seq2, *dev_matrix;
    int * dev_weights,*dev_score,*host_score;
    int seq1_len, seq2_len, offsets;        // initialize variables
    seq1_len = strlen(seq1);
    seq2_len = strlen(seq2);        // calculate the lengths of the two strings

    offsets = seq1_len- seq2_len;   // caculate the difference between the lengths , for the offset size

    int numthreads = (end_hyp-start_hyp)*offsets; // calculate the number of needed threads, for the mission as the size of hyphen * offset
    int threadsPerBlock = 256; // default size for the number of threds in each block
    int numBlocks = numthreads / threadsPerBlock; // calculate how many blocks we need for that task


    if(numthreads % threadsPerBlock != 0) // if we have a remainder,we will use one extra block
        numBlocks++;


    
    hipMalloc((void**)&dev_seq1,sizeof(char)*(seq1_len+1));
    hipMalloc((void**)&dev_seq2,sizeof(char)*(seq2_len+1));
    hipMalloc((void**)&dev_matrix,sizeof(char)*MATRIXSIZE*MATRIXSIZE);        // allocate memory on device space for all this data stractures
    hipMalloc((void**)&dev_weights,sizeof(int)*4);                             // for the kernel function to calculate the scores
    hipMalloc((void**)&dev_score,sizeof(int)*(numthreads));
    hipMemcpy((void*)(dev_seq1),seq1,(seq1_len+1)*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy((void*)(dev_seq2),seq2,(seq2_len+1)*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy((void*)(dev_matrix),matrix,MATRIXSIZE*MATRIXSIZE*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy((void*)(dev_weights),weights,4*sizeof(int),hipMemcpyHostToDevice);     // copying all the data to the device memory from the host

    host_score = (int*)malloc(sizeof(int)*(numthreads)); // allocate an array on host ,that store the results calcualted on the device

    kernelFunc<<<numBlocks,threadsPerBlock>>>(dev_seq1,dev_seq2,dev_matrix,dev_weights,dev_score,offsets,seq2_len,numthreads); //execute the kernel function
                                                                                                                                // with all the relevant arguments

    hipDeviceSynchronize(); // makes the cpu wait until all gpu threads will end their tasks
    hipMemcpy(host_score,dev_score,sizeof(int)*numthreads,hipMemcpyDeviceToHost);//transfer the data from the device array to the host


    for(int i = start_hyp; i < end_hyp;i++){ // find the maximum score from all the scores that the cuda calculate , and save it in the scoreArr
        for(int j = 0; j< offsets;j++)
        {
            if(host_score[(i-start_hyp)*offsets+j] > scoreArr[0]){
                scoreArr[0] = host_score[(i-start_hyp)*offsets+j];
                scoreArr[1] = j;
                scoreArr[2] = i;
            }
        }
    }

    
    hipFree(dev_seq1);
    hipFree(dev_seq2);
    hipFree(dev_matrix);      //releasing all the allocated space,of device and later of the host
    hipFree(dev_score);
    hipFree(dev_weights);
    free(host_score);

}